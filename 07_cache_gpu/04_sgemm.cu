
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipblas.h>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>
using namespace std;

#define M 1024

__global__ void matmul(float *A, float *B, float *C, int N) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  __shared__ float A_s[M];
  for (int ks=0; ks<N; ks+=M) {
    __syncthreads();
    A_s[threadIdx.x] = A[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+M; k++) {
      sum += A_s[k-ks] * B[N*k+j];
    }
  }
  C[N*i+j] = sum;
}

int main(int argc, char **argv) {
  int N = 2048;
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float * h_D = new float [N*N];
  float *d_A, *d_B, *d_C, *d_D;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);
  hipMalloc((void **) &d_D, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
      h_D[N*i+j] = 0;
    }
  }
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  dim3 grid(N/M, N);
  struct timeval tic, toc;
  gettimeofday(&tic, NULL);
  matmul<<<grid,M>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
  gettimeofday(&toc, NULL);
  double time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  stat = hipblasSetMatrix(N, N, sizeof(*h_A), h_A, N, d_A, N);
  stat = hipblasSetMatrix(N, N, sizeof(*h_B), h_B, N, d_B, N);
  stat = hipblasSetMatrix(N, N, sizeof(*h_D), h_D, N, d_D, N);
  gettimeofday(&tic, NULL);
  stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                     &alpha, d_B, N, d_A, N, &beta, d_D, N);
  stat = hipblasGetMatrix(N, N, sizeof(*h_D), d_D, N, h_D, N);
  gettimeofday(&toc, NULL);
  time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]-h_D[N*i+j]);
    }
  }
  printf("error: %f\n",err/N/N);
#pragma omp parallel for
  for (int i=0; i<N; i++)
    for (int k=0; k<N; k++)
      for (int j=0; j<N; j++)
        h_D[N*i+j] -= h_A[N*i+k] * h_B[N*k+j];
  err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(h_D[N*i+j]);
  printf("error: %lf\n",err/N/N);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);
  hipblasDestroy(handle);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_D;
}
