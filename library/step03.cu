
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipblas.h>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>

#define M 1024

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return double(tv.tv_sec)+double(tv.tv_usec)*1e-6;
}

__global__ void matmul(float *A, float *B, float *C, int N) {
  int i = blockIdx.y;
  int j = threadIdx.x + blockDim.x * blockIdx.x;
  float sum = 0.0f;
  __shared__ float s_A[M];
  for (int ks=0; ks<N; ks+=M) {
    __syncthreads();
    s_A[threadIdx.x] = A[N*i+ks+threadIdx.x];
    __syncthreads();
    for (int k=ks; k<ks+M; k++) {
      sum += s_A[k-ks] * B[N*k+j];
    }
  }
  C[N*i+j] = sum;
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float * h_D = new float [N*N];
  float *d_A, *d_B, *d_C, *d_D;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);
  hipMalloc((void **) &d_D, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
      h_D[N*i+j] = 0;
    }
  }
  double tic = get_time();
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  dim3 grid(N/M, N);
  matmul<<<grid,M>>>(d_A, d_B, d_C, N);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  double toc = get_time();
  printf("N=%d: %lf s (%lf GFlops)\n",N,toc-tic,2.*N*N*N/(toc-tic)/1e9);
  tic = get_time();
  float alpha = 1.0;
  float beta = 1.0;
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  stat = hipblasSetMatrix(N, N, sizeof(*h_A), h_A, N, d_A, N);
  stat = hipblasSetMatrix(N, N, sizeof(*h_B), h_B, N, d_B, N);
  stat = hipblasSetMatrix(N, N, sizeof(*h_D), h_D, N, d_D, N);
  stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N,
                     &alpha, d_A, N, d_B, N, &beta, d_D, N);
  stat = hipblasGetMatrix(N, N, sizeof(*h_D), d_D, N, h_D, N);
  toc = get_time();
  printf("N=%d: %lf s (%lf GFlops)\n",N,toc-tic,2.*N*N*N/(toc-tic)/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]-h_D[N*j+i]);
    }
  }
  printf("error: %f\n",err/N/N);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);
  hipblasDestroy(handle);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_D;
}
