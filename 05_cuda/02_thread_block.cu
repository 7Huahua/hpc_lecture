
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  a[i] = i;
}

int main(void) {
  int n = 2048;
  int size = n * sizeof(float);
  float *a, *b = (float*) malloc(size);
  hipMalloc(&a, size);
  kernel<<<n/1024,1024>>>(a);
  hipMemcpy(b, a, size, hipMemcpyDeviceToHost);
  for (int i=0; i<n; i++) printf("%f\n",b[i]);
  hipFree(a);
  free(b);
  return 0;
}