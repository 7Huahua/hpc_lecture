
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  a[i] = i;
}

int main(void) {
  int N = 2048;
  float *a;
  hipMallocManaged(&a, N*sizeof(float));
  kernel<<<N/1024,1024>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}