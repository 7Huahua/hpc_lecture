
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  a[threadIdx.x] = threadIdx.x;
}

int main(void) {
  int size = 4 * sizeof(float);
  float *a;
  hipMallocManaged(&a, size);
  kernel<<<1,4>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<4; i++) printf("%f\n",a[i]);
  hipFree(a);
  return 0;
}