
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  a[threadIdx.x] = threadIdx.x;
}

int main(void) {
  int n = 4;
  int size = n * sizeof(float);
  float *a, *b = (float*) malloc(size);
  hipMalloc(&a, size);
  kernel<<<1,n>>>(a);
  hipMemcpy(b, a, size, hipMemcpyDeviceToHost);
  for (int i=0; i<n; i++) printf("%f\n",b[i]);
  hipFree(a);
  free(b);
  return 0;
}