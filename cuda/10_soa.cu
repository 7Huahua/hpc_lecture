
#include <hip/hip_runtime.h>
#include <cstdio>

struct Record {
  int *key;
  int *value;
  int *flag;
};

__global__ void kernel(Record a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    a.key[i] = i;
}

int main(void) {
  int n = 2047;
  int m = 1024;
  int size = n * sizeof(int);
  Record a, b;
  b.key = (int*) malloc(size);
  b.value = (int*) malloc(size);
  b.flag = (int*) malloc(size);
  hipMalloc(&a.key, size);
  hipMalloc(&a.value, size);
  hipMalloc(&a.flag, size);
  kernel<<<(n+m-1)/m,m>>>(a,n);
  hipMemcpy(b.key, a.key, size, hipMemcpyDeviceToHost);
  hipMemcpy(b.value, a.value, size, hipMemcpyDeviceToHost);
  hipMemcpy(b.flag, a.flag, size, hipMemcpyDeviceToHost);
  for (int i=0; i<n; i++) printf("%d\n",b.key[i]);
  hipFree(a.key);
  hipFree(a.value);
  hipFree(a.flag);
  free(b.key);
  free(b.value);
  free(b.flag);
  return 0;
}