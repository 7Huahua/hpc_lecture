
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  a[i] = 10 * blockIdx.x + threadIdx.x;
}

int main(void) {
  int size = 4 * sizeof(float);
  float *a;
  hipMallocManaged(&a, size);
  kernel<<<2,2>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<4; i++) printf("%f\n",a[i]);
  hipFree(a);
  return 0;
}