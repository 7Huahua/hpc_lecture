
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  if (i % 2)
    a[i] = i;
  else
    a[i] = -i;
}

int main(void) {
  int n = 2047;
  int m = 1024;
  int size = n * sizeof(float);
  float *a, *b = (float*) malloc(size);
  hipMalloc(&a, size);
  kernel<<<(n+m-1)/m,m>>>(a,n);
  hipMemcpy(b, a, size, hipMemcpyDeviceToHost);
  for (int i=0; i<n; i++) printf("%f\n",b[i]);
  hipFree(a);
  free(b);
  return 0;
}