
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    a[i] = i;
}

int main(void) {
  int n = 2047;
  int m = 1024;
  int size = n * sizeof(float);
  float *a;
  hipMallocManaged(&a, size);
  kernel<<<(n+m-1)/m,m>>>(a,n);
  hipDeviceSynchronize();
  for (int i=0; i<n; i++) printf("%f\n",a[i]);
  hipFree(a);
  return 0;
}