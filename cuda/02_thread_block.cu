
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  a[i] = 10 * blockIdx.x + threadIdx.x;
}

int main(void) {
  const int size = 4 * sizeof(float);
  float *a, *b = (float*) malloc(size);
  hipMalloc(&a, size);
  kernel<<<2,2>>>(a);
  hipMemcpy(b, a, size, hipMemcpyDeviceToHost);
  for (int i=0; i<4; i++) printf("%f\n",b[i]);
  hipFree(a);
  free(b);
  return 0;
}