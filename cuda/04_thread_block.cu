
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  a[i] = i;
}

int main(void) {
  int n = 2048;
  int size = n * sizeof(float);
  float *a;
  hipMallocManaged(&a, size);
  kernel<<<n/1024,1024>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<n; i++) printf("%f\n",a[i]);
  hipFree(a);
  return 0;
}