
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
    c[index] = a[index] + b[index];
}

#define N (2048*2048+43)
#define M 512

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  hipMalloc((void **) &d_a, size);
  hipMalloc((void **) &d_b, size);
  hipMalloc((void **) &d_c, size);

  a = (int *)malloc(size);
  b = (int *)malloc(size);
  c = (int *)malloc(size);
  for (int i=0; i<N; i++) {
    a[i] = 2;
    b[i] = 7;
  }

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, N);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  int sum_c = 0;
  for (int i=0; i<N; i++)
    sum_c += c[i];
  printf("%d %d\n",sum_c,N*9);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}