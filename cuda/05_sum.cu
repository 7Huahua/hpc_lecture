
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= n) return;
  a[0] += a[i];
  //atomicAdd(&a[0], a[i]);
}

int main(void) {
  int n = 2047;
  int m = 1024;
  int size = n * sizeof(float);
  float *a, *b = (float*) malloc(size);
  hipMalloc(&a, size);
  for (int i=0; i<n; i++) b[i] = 1;
  hipMemcpy(a, b, size, hipMemcpyHostToDevice);
  kernel<<<(n+m-1)/m,m>>>(a, n);
  hipMemcpy(b, a, size, hipMemcpyDeviceToHost);
  printf("%f\n",b[0]);
  hipFree(a);
  free(b);
  return 0;
}
