
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a, float *sum) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //*sum += a[i];
  atomicAdd(sum, a[i]);
}

int main(void) {
  int size = 4 * sizeof(float);
  float *a, *sum;
  hipMallocManaged(&a, size);
  hipMallocManaged(&sum, sizeof(float));
  for (int i=0; i<4; i++) a[i] = 1;
  kernel<<<2,2>>>(a, sum);
  hipDeviceSynchronize();
  printf("%f\n",*sum);
  hipFree(a);
  hipFree(sum);
  return 0;
}